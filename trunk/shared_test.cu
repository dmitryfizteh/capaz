#include "gpu.h"
#include "cuPrintf.cu"

// �������� ������ GPU
void checkErrors(char *label, char *file, int line) 
{
#ifdef MY_TEST
	hipError_t err;

	err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		char *e = (char*) hipGetErrorString(err);
		printf("CUDA Error: %s (at %s)\nFile:\"%s\"\nLine:\"%d\"\n\n", e, label, file, line);
	}

	err=hipGetLastError();
	if (err != hipSuccess)
	{
		char *e = (char*) hipGetErrorString(err);
		printf("CUDA Error: %s (at %s)\nFile:\"%s\"\nLine:\"%d\"\n\n", e, label, file, line);
		fflush(stdout);
	}
#endif
}

// ���� �� NaN
// ��������� ������ device_test_nan(x, __FILE__, __LINE__);
__device__ void device_test_nan (double x, char *file, int line)
{
#ifdef MY_TEST
	if ( (x>1e+30) || (x<-1*1e+40))
		CUPRINTF("Error: NaN\nFile:\"%s\"\nLine:\"%d\"\n\n", file, line);
#endif
}

// ���� �� ������������� � �� NaN
// ��������� ������ device_test_positive(x, __FILE__, __LINE__);
__device__ void device_test_positive (double x, char *file, int line)
{
#ifdef MY_TEST
	if ( (x>1e+30) || (x<0))
		CUPRINTF("Error: NaN or X<0\nFile:\"%s\"\nLine:\"%d\"\n\n", file, line);
#endif
}

// ���� �� ��������� ������������� � [0;1]
// ��������� ������ device_test_S(x, __FILE__, __LINE__);
__device__ void device_test_S (double S, char *file, int line)
{
#ifdef MY_TEST
	if ( S < 0 )
		CUPRINTF("Error: S<0\nFile:\"%s\"\nLine:\"%d\"\n\n", file, line);
	if ( S > 1 )
		CUPRINTF("Error: S>1\nFile:\"%s\"\nLine:\"%d\"\n\n", file, line);
#endif
}