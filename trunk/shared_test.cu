#include "gpu.h"
#include "cuPrintf.cu"

// �������� ������ GPU
void checkErrors(char *label) 
{
#ifdef MY_TEST
	hipError_t err;

	err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		char *e = (char*) hipGetErrorString(err);
		printf("CUDA Error: %s (at %s)\n", e, label);
	}

	err=hipGetLastError();
	if (err != hipSuccess)
	{
		char *e = (char*) hipGetErrorString(err);
		printf("CUDA Error: %s (at %s)\n", e, label);
		fflush(stdout);
	}
#endif
}

// ���� �� NaN
// ��������� ������ test_nan(x, __FILE__, __LINE__);
__device__ void device_test_nan (double x, char *file, int line)
{
#ifdef MY_TEST
	if ( (x>1e+30) || (x<-1*1e+40))
		CUPRINTF("Error: NaN\nFile:\"%s\"\nLine:\"%d\"\n\n", file, line);
#endif
}

// ���� �� ������������� � �� NaN
// ��������� ������ test_nan(x, __FILE__, __LINE__);
__device__ void device_test_positive (double x, char *file, int line)
{
#ifdef MY_TEST
	if ( (x>1e+30) || (x<0))
		CUPRINTF("Error: NaN or X<0\nFile:\"%s\"\nLine:\"%d\"\n\n", file, line);
#endif
}