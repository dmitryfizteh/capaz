#include "hip/hip_runtime.h"
//#include "../defines.h"
#include "../gpu.h"
#include "b-l.h"

//****************************
//TODO: copy to another place
//****************************

// �������� �� ����������� ��������
double Injection_well_P(ptr_Arrays HostArraysPtr, int i, int j, int k, consts def)
{
	// 10000psi in Pa
	return INJECTION_WELL_Pw; //68947572.9;
}

// �������� �� ���������� ��������
double Production_well_P(ptr_Arrays HostArraysPtr, int i, int j, int k, consts def)
{
	// 4000psi in Pa
	return OUTPUT_WELL_Pw;//27579029.16;
}

// ���������� ��������� �������
void data_initialization(ptr_Arrays HostArraysPtr, long int* t, consts def)
{
	*t = 0;
	for (int i = 0; i < def.locNx; i++)
		for (int j = 0; j < def.locNy; j++)
			for (int k = 0; k < def.locNz; k++)
				if (is_active_point(i, j, k, def))
				{
					// �������������� ��������� ��������� ���������� � ����������
					int I = local_to_global(i, 'x', def);

					//HostArraysPtr.m[i + j * def.locNx + k * def.locNx * def.locNy] = 0;
					HostArraysPtr.S_n[i + j * def.locNx + k * def.locNx * def.locNy] = BACKGROUND_Sn;
					//HostArraysPtr.S_n[i+j*def.locNx+k*def.locNx*def.locNy] =0.3 + 0.3 * j / def.Ny;

					double ro_g_dy = (def.ro0_n * HostArraysPtr.S_n[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)]
					                  + def.ro0_w * (1 - HostArraysPtr.S_n[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)])) * (def.m[i + j * def.locNx + k * def.locNx * def.locNy]) * (def.g_const) * (def.hy);

					// 6000 pound per square inch = 41 368 543.8 �������
					if (j == 0)
					{
						HostArraysPtr.P_w[i + j * def.locNx + k * def.locNx * def.locNy] = BACKGROUND_Pw; //+ 5000000;    // 50368543.8;
					}
					else
					{
						HostArraysPtr.P_w[i + j * def.locNx + k * def.locNx * def.locNy] = HostArraysPtr.P_w[i + (j - 1) * def.locNx + k * def.locNx * def.locNy] + ro_g_dy;
					}

					
					// � ������ ���������� ��������� ����������� ��������
					if ((i == def.Nx / 2) && (j == def.Ny / 2) && (k == def.Nz / 2))
					{
						HostArraysPtr.P_w[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] = Injection_well_P(HostArraysPtr, i, j, k, def);
						//HostArraysPtr.S_n[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] = 0.5;
					}

					// � ������ ���������� ��������� ���������� ��������
					if ((i == def.Nx - 3) && (j == def.Ny / 2) && (k == def.Nz - 3))
					{
						HostArraysPtr.P_w[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] = Production_well_P(HostArraysPtr, i, j, k, def);
						//HostArraysPtr.S_n[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] = 0.5;
					}
					

					HostArraysPtr.ro_w[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] = def.ro0_w * (1. + (def.beta_w) * (HostArraysPtr.P_w[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] - def.P_atm));
					HostArraysPtr.ro_n[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] = def.ro0_n * (1. + (def.beta_n) * (HostArraysPtr.P_w[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] - def.P_atm));

					test_nan(HostArraysPtr.S_n[i + j * def.locNx + k * def.locNx * def.locNy], __FILE__, __LINE__);
					test_nan(HostArraysPtr.P_w[i + j * def.locNx + k * def.locNx * def.locNy], __FILE__, __LINE__);
					test_nan(HostArraysPtr.m[i + j * def.locNx + k * def.locNx * def.locNy], __FILE__, __LINE__);
				}
}

//****************************
//TODO: copy to another place
//****************************


// ������ ����������, �������� NAPL P2 � Xi � ������ ����� ����� (���������� �� ��������� �����)
__global__ void assign_ro_Pn_Xi_kernel(ptr_Arrays DevArraysPtr)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if ((i < ((*gpu_def).locNx)) && (j < ((*gpu_def).locNy)) && (k < ((*gpu_def).locNz)) && (device_is_active_point(i, j, k) == 1))
	{
		double k_w, k_n;
		double S = 1 - DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)];

		double S_wc = 0.2;
		//		double S_wi=0.2;
		double S_or = 0.2;
		double S_e = (S - S_wc) / (1 - S_wc - S_or);

		k_w = S_e * S_e;
		k_n = (1 - S_e) * (1 - S_e);

		//krw(Sor) = kro(Swc) = 1.0

		if (S < S_wc)
		{
			k_w = 0;
			k_n = 1;
		}

		if (S > (1 - S_or))
		{
			k_w = 1;
			k_n = 0;
		}

		DevArraysPtr.P_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)];
		DevArraysPtr.Xi_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = -1 * (DevArraysPtr.K[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)]) * k_w / (*gpu_def).mu_w;
		DevArraysPtr.Xi_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = -1 * (DevArraysPtr.K[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)]) * k_n / (*gpu_def).mu_n;
		DevArraysPtr.ro_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = (*gpu_def).ro0_w * (1 + ((*gpu_def).beta_w) * (DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] - (*gpu_def).P_atm));
		DevArraysPtr.ro_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = (*gpu_def).ro0_n * (1 + ((*gpu_def).beta_n) * (DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] - (*gpu_def).P_atm));

		device_test_positive(DevArraysPtr.P_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr.ro_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr.ro_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
		device_test_nan(DevArraysPtr.Xi_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
		device_test_nan(DevArraysPtr.Xi_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
	}
}


// ����� ������� ��� ������ ����� ����� (���������� �� ��������� �����)
__global__ void Newton_method_kernel(ptr_Arrays DevArraysPtr)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if ((i < ((*gpu_def).locNx) - 1) && (j < (*gpu_def).locNy - 1) && (k < ((*gpu_def).locNz)) && (i != 0) && (j != 0) && (((k != 0) && (k != ((*gpu_def).locNz) - 1)) || (((*gpu_def).locNz) < 2)))
	{
		double A1 = DevArraysPtr.roS_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)];
		double A2 = DevArraysPtr.roS_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)];
		double a = (*gpu_def).beta_w * ((*gpu_def).beta_n);
		double b = (*gpu_def).beta_w + (*gpu_def).beta_n - A2 * ((*gpu_def).beta_w) / ((*gpu_def).ro0_n) - A1 * ((*gpu_def).beta_n) / ((*gpu_def).ro0_w);
		double c = 1 - A2 / (*gpu_def).ro0_n  - A1 / (*gpu_def).ro0_w;
		double D = b * b - 4 * a * c;
		double P1 = (*gpu_def).P_atm + (-1 * b + sqrt(D)) / (2 * a);
		double P2 = (*gpu_def).P_atm + (-1 * b - sqrt(D)) / (2 * a);

		if (P1 < 0)
		{
			DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = P2;
		}
		else
		{
			DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = P1;
		}

		DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = DevArraysPtr.roS_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] / ((*gpu_def).ro0_n * (1 + (*gpu_def).beta_n * (DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] - (*gpu_def).P_atm)));

		if (DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] < 0.2)
		{
			DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = 0.2;
		}

		if (DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] > 0.8)
		{
			DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = 0.8;
		}

		/*
		double F1, F2, F1P, F2P, F1S, F2S, det;
		double S_n=DevArraysPtr.S_n[i+j*((*gpu_def).locNx)+k*((*gpu_def).locNx)*((*gpu_def).locNy)];
		double P_w=DevArraysPtr.P_w[i+j*((*gpu_def).locNx)+k*((*gpu_def).locNx)*((*gpu_def).locNy)];

		//S_e = (1 - S_n - (*gpu_def).S_wr[media]) / (1 - (*gpu_def).S_wr[media]);

		//AAA = pow(S_e, ((-1 / (*gpu_def).lambda[media]) - 1));
		F1 = (*gpu_def).ro0_w * (1 + ((*gpu_def).beta_w) * (P_w - (*gpu_def).P_atm)) * (1 - S_n) - DevArraysPtr.roS_w[i+j*((*gpu_def).locNx)+k*((*gpu_def).locNx)*((*gpu_def).locNy)];
		F2 = (*gpu_def).ro0_n * (1 + ((*gpu_def).beta_n) * (P_w - (*gpu_def).P_atm)) * S_n - DevArraysPtr.roS_n[i+j*((*gpu_def).locNx)+k*((*gpu_def).locNx)*((*gpu_def).locNy)];

		F1P = (*gpu_def).ro0_w * ((*gpu_def).beta_w) * (1 - S_n);
		F2P = (*gpu_def).ro0_n * ((*gpu_def).beta_n) * S_n;
		F1S = (-1) * (*gpu_def).ro0_w * (1 + ((*gpu_def).beta_w) * (P_w - (*gpu_def).P_atm));
		F2S = (*gpu_def).ro0_n * (1 + ((*gpu_def).beta_n) * (P_w - (*gpu_def).P_atm));

		det = F1P * F2S - F1S * F2P;

		DevArraysPtr.P_w[i+j*((*gpu_def).locNx)+k*((*gpu_def).locNx)*((*gpu_def).locNy)] = P_w - (1 / det) * (F2S * F1 - F1S * F2);
		DevArraysPtr.S_n[i+j*((*gpu_def).locNx)+k*((*gpu_def).locNx)*((*gpu_def).locNy)] = S_n - (1 / det) * (F1P * F2 - F2P * F1);
		*/

		device_test_positive(DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
		device_test_S(DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
	}
}

// �������� �� �������������� ��������
__device__ double device_Injection_well_P(ptr_Arrays DevArraysPtr, int i, int j, int k)
{
	// 10000psi in Pa
	return INJECTION_WELL_Pw;//(*gpu_def).P_atm + 10000000; //68947572.9;
}

// �������� �� ���������� ��������
__device__ double device_Production_well_P(ptr_Arrays DevArraysPtr, int i, int j, int k)
{
	// 4000psi in Pa
	return OUTPUT_WELL_Pw;//(*gpu_def).P_atm;//27579029.16;
}

// ���������� ��������� ������� ��� ������ ����� ����� (���������� �� ��������� �����)
__global__ void data_initialization_kernel(ptr_Arrays DevArraysPtr, long int* t)
{
	*t = 0;
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;
	if (device_is_active_point(i, j, k))
	{
		// �������������� ��������� ��������� ���������� � ����������
		int I = device_local_to_global(i, 'x');

		//DevArraysPtr.m[i + j * (*gpu_def).locNx + k * (*gpu_def).locNx * (*gpu_def).locNy] = 0;
		DevArraysPtr.S_n[i + j * (*gpu_def).locNx + k * (*gpu_def).locNx * (*gpu_def).locNy] = 0.7;
		//DevArraysPtr.S_n[i+j*(*gpu_def).locNx+k*(*gpu_def).locNx*(*gpu_def).locNy] =0.3 + 0.3 * j / (*gpu_def).Ny;

		double ro_g_dy = ((*gpu_def).ro0_n * DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)]
		                  + (*gpu_def).ro0_w * (1 - DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)])) * ((*gpu_def).m[i + j * (*gpu_def).locNx + k * (*gpu_def).locNx * (*gpu_def).locNy]) * ((*gpu_def).g_const) * ((*gpu_def).hy);

		// 6000 pound per square inch = 41 368 543.8 �������
		if (j == 0)
		{
			DevArraysPtr.P_w[i + j * (*gpu_def).locNx + k * (*gpu_def).locNx * (*gpu_def).locNy] = (*gpu_def).P_atm; //+ 5000000;    // 50368543.8;
		}
		else
		{
			DevArraysPtr.P_w[i + j * (*gpu_def).locNx + k * (*gpu_def).locNx * (*gpu_def).locNy] = DevArraysPtr.P_w[i + (j - 1) * (*gpu_def).locNx + k * (*gpu_def).locNx * (*gpu_def).locNy] + ro_g_dy;
		}

		/*
		// � ������ ���������� ��������� �������������� ��������
		if ((i == (*gpu_def).Nx / 2) && (j == (*gpu_def).Ny / 2) && (k == (*gpu_def).Nz / 2))
		{
			DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = device_Injection_well_P(DevArraysPtr, i, j, k);
			//DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = 0.5;
		}

		// � ������ ���������� ��������� ���������� ��������
		if ((i == (*gpu_def).Nx - 3) && (j == (*gpu_def).Ny / 2) && (k == (*gpu_def).Nz - 3))
		{
			DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = device_Production_well_P(DevArraysPtr, i, j, k);
			//DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = 0.5;
		}
		*/

		DevArraysPtr.ro_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = (*gpu_def).ro0_w * (1. + ((*gpu_def).beta_w) * (DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] - (*gpu_def).P_atm));
		DevArraysPtr.ro_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = (*gpu_def).ro0_n * (1. + ((*gpu_def).beta_n) * (DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] - (*gpu_def).P_atm));

		device_test_nan(DevArraysPtr.S_n[i + j * (*gpu_def).locNx + k * (*gpu_def).locNx * (*gpu_def).locNy], __FILE__, __LINE__);
		device_test_nan(DevArraysPtr.P_w[i + j * (*gpu_def).locNx + k * (*gpu_def).locNx * (*gpu_def).locNy], __FILE__, __LINE__);
		device_test_nan(DevArraysPtr.m[i + j * (*gpu_def).locNx + k * (*gpu_def).locNx * (*gpu_def).locNy], __FILE__, __LINE__);
	}
}

