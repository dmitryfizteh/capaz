#include "hip/hip_runtime.h"
//#include "../defines.h"
#include "../gpu.h"
#include "b-l.h"

//****************************
//TODO: copy to another place
//****************************

// ���������� ��������� �������
void data_initialization(ptr_Arrays HostArraysPtr, long int* t, consts def)
{
	*t = 0;
	for (int i = 0; i < def.locNx; i++)
		for (int j = 0; j < def.locNy; j++)
			for (int k = 0; k < def.locNz; k++)
				if (is_active_point(i, j, k, def))
				{
					HostArraysPtr.m[i + j * def.locNx + k * def.locNx * def.locNy]=def.m[0];
					HostArraysPtr.S_n[i + j * def.locNx + k * def.locNx * def.locNy] = def.Background_Sn;

					double ro_g_dy = (def.ro0_n * HostArraysPtr.S_n[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)]
					                  + def.ro0_w * (1 - HostArraysPtr.S_n[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)])) * (HostArraysPtr.m[i + j * def.locNx + k * def.locNx * def.locNy]) * (def.g_const) * (def.hy);

					if (j == 0)
					{
						HostArraysPtr.P_w[i + j * def.locNx + k * def.locNx * def.locNy] = def.P_atm;
					}
					else
					{
						HostArraysPtr.P_w[i + j * def.locNx + k * def.locNx * def.locNy] = HostArraysPtr.P_w[i + (j - 1) * def.locNx + k * def.locNx * def.locNy] + ro_g_dy;
					}

					/*
					// �������������� ��������
					if (is_injection_well(i, j, k, def))
					{
						HostArraysPtr.P_w[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] = Injection_well_P(HostArraysPtr, i, j, k, def);
						//HostArraysPtr.S_n[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] = 0.5;
					}

					// ���������� ��������
					if (is_output_well(i, j, k, def))
					{
						HostArraysPtr.P_w[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] = Production_well_P(HostArraysPtr, i, j, k, def);
						//HostArraysPtr.S_n[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] = 0.5;
					}
					*/

					HostArraysPtr.ro_w[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] = def.ro0_w * (1. + (def.beta_w) * (HostArraysPtr.P_w[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] - def.P_atm));
					HostArraysPtr.ro_n[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] = def.ro0_n * (1. + (def.beta_n) * (HostArraysPtr.P_w[i + j * (def.locNx) + k * (def.locNx) * (def.locNy)] - def.P_atm));

					test_S(HostArraysPtr.S_n[i + j * def.locNx + k * def.locNx * def.locNy], __FILE__, __LINE__);
					test_positive(HostArraysPtr.P_w[i + j * def.locNx + k * def.locNx * def.locNy], __FILE__, __LINE__);
					test_positive(HostArraysPtr.m[i + j * def.locNx + k * def.locNx * def.locNy], __FILE__, __LINE__);
				}
}
//****************************
//TODO: copy to another place
//****************************

// ������ ������������� �������������� � �����
__device__ void assing_k(double* k_w, double* k_n, double S_w)
{
	/*
	// SPE-����������
	double S_wc = 0.2;
	double S_or = 0.2;
	double S_e = (S_w - S_wc) / (1. - S_wc - S_or);

	*k_w = S_e * S_e;
	*k_n = (1. - S_e) * (1. - S_e);

	if (S_w < S_wc)
	{
		*k_w = 0.;
		*k_n = 1.;
	}

	if (S_w > (1 - S_or))
	{
		*k_w = 1.;
		*k_n = 0.;
	}
	*/
	
	// ���������� ���
	double S_sv = 0.1;
	double S_zv = 0.8;
	double S_1 = 0.70324;

	if ((S_sv<=S_w) && (S_w<=S_zv))
		*k_n=((S_zv-S_w)/(S_zv-S_sv))*((S_zv-S_w)/(S_zv-S_sv));
	else
		if ((0<=S_w) && (S_w<=S_sv))
			*k_n=1.;
		else //S_zv<S<=1
			*k_n=0.;

	if ((S_sv<=S_w) && (S_w<=S_1))
		*k_w=((S_w-S_sv)/(S_zv-S_sv))*((S_w-S_sv)/(S_zv-S_sv));
	else
		if ((0<=S_w) && (S_w<=S_sv))
			*k_w=0.;
		else 
			if ((S_1<=S_w) && (S_w<=S_zv))
				*k_w=0.8*pow((S_w-S_sv)/(S_zv-S_sv), 0.5);
			else//S_zv<S<=1
				*k_w=1.;

	device_test_S(*k_n, __FILE__, __LINE__);
	device_test_S(*k_w, __FILE__, __LINE__);
}

// ������ ����������, �������� NAPL P2 � Xi � ������ ����� ����� (���������� �� ��������� �����)
__global__ void assign_ro_Pn_Xi_kernel(ptr_Arrays DevArraysPtr)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if ((i < ((*gpu_def).locNx)) && (j < ((*gpu_def).locNy)) && (k < ((*gpu_def).locNz)) && (device_is_active_point(i, j, k) == 1))
	{
		double k_w=0., k_n=0.;
		assing_k(&k_w, &k_n, 1. - DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)]);

		DevArraysPtr.P_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)];
		DevArraysPtr.Xi_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = -1 * (DevArraysPtr.K[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)]) * k_w / (*gpu_def).mu_w;
		DevArraysPtr.Xi_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = -1 * (DevArraysPtr.K[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)]) * k_n / (*gpu_def).mu_n;
		DevArraysPtr.ro_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = (*gpu_def).ro0_w * (1 + ((*gpu_def).beta_w) * (DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] - (*gpu_def).P_atm));
		DevArraysPtr.ro_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = (*gpu_def).ro0_n * (1 + ((*gpu_def).beta_n) * (DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] - (*gpu_def).P_atm));

		device_test_positive(DevArraysPtr.P_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr.ro_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr.ro_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
		device_test_nan(DevArraysPtr.Xi_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
		device_test_nan(DevArraysPtr.Xi_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
	}
}


// ����� ������� ��� ������ ����� ����� (���������� �� ��������� �����)
__global__ void Newton_method_kernel(ptr_Arrays DevArraysPtr)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if ((i < ((*gpu_def).locNx) - 1) && (j < (*gpu_def).locNy - 1) && (k < ((*gpu_def).locNz)) && (i != 0) && (j != 0) && (((k != 0) && (k != ((*gpu_def).locNz) - 1)) || (((*gpu_def).locNz) < 2)))
	{
		double A1 = DevArraysPtr.roS_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)];
		double A2 = DevArraysPtr.roS_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)];
		double a = (*gpu_def).beta_w * ((*gpu_def).beta_n);
		double b = (*gpu_def).beta_w + (*gpu_def).beta_n - A2 * ((*gpu_def).beta_w) / ((*gpu_def).ro0_n) - A1 * ((*gpu_def).beta_n) / ((*gpu_def).ro0_w);
		double c = 1. - A2 / (*gpu_def).ro0_n  - A1 / (*gpu_def).ro0_w;
		double D = b * b - 4. * a * c;
		double P1 = (*gpu_def).P_atm + (-1. * b + sqrt(D)) / (2. * a);
		double P2 = (*gpu_def).P_atm + (-1. * b - sqrt(D)) / (2. * a);

		if (P1 < 0.)
		{
			DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = P2;
		}
		else
		{
			DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = P1;
		}

		DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] = DevArraysPtr.roS_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] / ((*gpu_def).ro0_n * (1 + (*gpu_def).beta_n * (DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)] - (*gpu_def).P_atm)));

		device_test_positive(DevArraysPtr.P_w[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
		device_test_S(DevArraysPtr.S_n[i + j * ((*gpu_def).locNx) + k * ((*gpu_def).locNx) * ((*gpu_def).locNy)], __FILE__, __LINE__);
	}
}

