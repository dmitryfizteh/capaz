#include "hip/hip_runtime.h"
#include "gpu.h"
#include "shared_test.cu"

__constant__ consts gpu_def [1];

// �������������� ��������� ��������� ���������� � ����������
// ������ ��������� �������� �������������� ����� � ������� ���
// ������ �������, ���� ����� ������ 
// (���� 2 ������ � ����� ������,�� +2 �����). 
// ���������� ������� �������� ��� ������� ����� (������ � ������� �� rank==0)
__device__ int device_i_to_I(int i, int rank, int size, consts def)
{
	int I;
	if (rank <= ((*gpu_def).Nx)%size)
	{
		if(rank==0)
			I=i;
		else
			I=(((*gpu_def).Nx)/size+1)*rank+i-1;
	}
	else
		I=(((*gpu_def).Nx)/size+1)*rank-(rank-((*gpu_def).Nx)%size)+i-1;
	return I;
}

// �������� �� ����� �������� (�.�. �� ��������������� ������ ��� ������ �� ��������)
__device__ int device_is_active_point(int i, int localNx, int rank, int size)
{
	if((rank!=0 && i==0) || (rank!=size-1 && i==localNx-1))
		return 0;
	else
		return 1;
}

//========================================================================================================================
// ����� ���������� ��������� � ��������� ����� ��������. �� ���� ����������.

// ������ ����������, �������� NAPL P2 � Xi � ������ ����� ����� (���������� �� ��������� �����)
__global__ void assign_ro_Pn_Xi_kernel(ptr_Arrays DevArraysPtr, int localNx, int rank, int size) 
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int j=threadIdx.y+blockIdx.y*blockDim.y;
	int k=threadIdx.z+blockIdx.z*blockDim.z;

	if ((i<localNx) && (j<((*gpu_def).Ny)) && (k<((*gpu_def).Nz)) && (device_is_active_point(i, localNx, rank, size)==1))
	{
		int media = DevArraysPtr.media[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double S_n = DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P_w = DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		double S_e = (1.- S_n - (*gpu_def).S_wr[media]) / (1. - (*gpu_def).S_wr[media]);
		double k_w = pow(S_e, (2. + 3. * (*gpu_def).lambda[media]) / (*gpu_def).lambda[media]);
		double k_n = (1. - S_e) * (1. - S_e) * (1 - pow(S_e, (2. + (*gpu_def).lambda[media]) / (*gpu_def).lambda[media]));
		double P_k = (*gpu_def).P_d[media] * pow((1. - S_n - (*gpu_def).S_wr[media]) / (1. - (*gpu_def).S_wr[media]), -1. / (*gpu_def).lambda[media]);

		DevArraysPtr.P_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = P_w + P_k;
		DevArraysPtr.Xi_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = -1 * (*gpu_def).K[media] * k_w / (*gpu_def).mu_w;
		DevArraysPtr.Xi_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = -1 * (*gpu_def).K[media] * k_n / (*gpu_def).mu_n;
		DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = (*gpu_def).ro0_w * (1 + ((*gpu_def).beta_w) * (P_w - (*gpu_def).P_atm));
		DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = (*gpu_def).ro0_n * (1 + ((*gpu_def).beta_n) * (P_w + P_k - (*gpu_def).P_atm));

		device_test_positive(DevArraysPtr.P_n[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
		device_test_nan(DevArraysPtr.Xi_w[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
		device_test_nan(DevArraysPtr.Xi_n[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
	}
}

// ������ ����������, �������� NAPL P2 � Xi �� ���� ������ �����
void ro_P_Xi_calculation(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, consts def, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ)
{
	assign_ro_Pn_Xi_kernel<<<dim3(blocksX,blocksY,blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,rank,size); 
	checkErrors("assign Pn, Xi, ro");
	cudaPrintfDisplay(stdout, true);
}

// ����� ������� ��� ������ ����� ����� (���������� �� ��������� �����)
__global__ void Newton_method_kernel(ptr_Arrays DevArraysPtr, int localNx) 
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int j=threadIdx.y+blockIdx.y*blockDim.y;
	int k=threadIdx.z+blockIdx.z*blockDim.z;

	if ((i<localNx-1) && (j<(*gpu_def).Ny-1) && (k<(*gpu_def).Nz) && (i!=0) && (j!=0) && (((k!=0) && (k!=(*gpu_def).Nz-1)) || ((*gpu_def).Nz<2)))
	{
		int media = DevArraysPtr.media[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double S_e, P_k, AAA, F1, F2, PkS, F1P, F2P, F1S, F2S, det;
		double S_n=DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P_w=DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		S_e = (1 - S_n - (*gpu_def).S_wr[media]) / (1 - (*gpu_def).S_wr[media]);
		P_k = (*gpu_def).P_d[media] * pow(S_e, -1 / (*gpu_def).lambda[media]);
		AAA = pow(S_e, ((-1 / (*gpu_def).lambda[media]) - 1));
		F1 = (*gpu_def).ro0_w * (1 + ((*gpu_def).beta_w) * (P_w - (*gpu_def).P_atm)) * (1 - S_n) - DevArraysPtr.roS_w[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		F2 = (*gpu_def).ro0_n * (1 + ((*gpu_def).beta_n) * (P_w + P_k - (*gpu_def).P_atm)) * S_n - DevArraysPtr.roS_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		PkS = AAA * (*gpu_def).P_d[media] / ((*gpu_def).lambda[media] * (1 - (*gpu_def).S_wr[media]));
		F1P = (*gpu_def).ro0_w * ((*gpu_def).beta_w) * (1 - S_n);
		F2P = (*gpu_def).ro0_n * ((*gpu_def).beta_n) * S_n;
		F1S = (-1) * (*gpu_def).ro0_w * (1 + ((*gpu_def).beta_w) * (P_w - (*gpu_def).P_atm));
		F2S = (*gpu_def).ro0_n * (1 + ((*gpu_def).beta_n) * (P_w + P_k - (*gpu_def).P_atm + (S_n * PkS)));

		det = F1P * F2S - F1S * F2P;

		DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = P_w - (1 / det) * (F2S * F1 - F1S * F2);
		DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = S_n - (1 / det) * (F1P * F2 - F2P * F1);

		device_test_positive(DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
	}
}

// ������ �������� ���� P1 � ������������ NAPL S2 �� ���� ������ �����
void P_S_calculation(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, consts def, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ)
{
	for (int w=1;w<=def.newton_iterations;w++)
	{
		Newton_method_kernel<<<dim3(blocksX,blocksY,blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx); 
		checkErrors("assign Pw and Sn");
		cudaPrintfDisplay(stdout, true);
	}
}

//========================================================================================================================

// ������ �������� � ������ ����� �����
__global__ void assign_u_kernel(ptr_Arrays DevArraysPtr, int localNx, int rank, int size) 
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int j=threadIdx.y+blockIdx.y*blockDim.y;
	int k=threadIdx.z+blockIdx.z*blockDim.z;

	if ((i<localNx) && (j<((*gpu_def).Ny)) && (k<((*gpu_def).Nz)) && (device_is_active_point(i, localNx, rank, size)==1))
	{
		//CUPRINTF("assign u\n");
		double Xi_w = DevArraysPtr.Xi_w[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double Xi_n = DevArraysPtr.Xi_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P_w = DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P_n = DevArraysPtr.P_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		if (((*gpu_def).Nx)>2)
		{
			if (i == 0)
			{
				DevArraysPtr.ux_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * (DevArraysPtr.P_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - P_w) / ((*gpu_def).hx);
				DevArraysPtr.ux_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * (DevArraysPtr.P_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - P_n) / ((*gpu_def).hx);
				//CUPRINTF("assign u=%e\n",DevArraysPtr.ux_w[i+j*localNx+k*localNx*((*gpu_def).Ny)]);
			}
			if (i == localNx - 1)
			{
				DevArraysPtr.ux_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * (P_w - DevArraysPtr.P_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / ((*gpu_def).hx);
				DevArraysPtr.ux_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * (P_n - DevArraysPtr.P_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / ((*gpu_def).hx);
			}
			if ((i != 0) && (i != localNx - 1))
			{
				DevArraysPtr.ux_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * ((DevArraysPtr.P_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.P_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hx)));
				DevArraysPtr.ux_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * ((DevArraysPtr.P_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.P_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hx)));
			}
		}
		else
		{
			DevArraysPtr.ux_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
			DevArraysPtr.ux_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
		}
	
		if (((*gpu_def).Ny)>2)
		{
			if (j == 0)
			{
				DevArraysPtr.uy_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * (
					(DevArraysPtr.P_w[i+(j+1)*localNx+k*localNx*(*gpu_def).Ny] - DevArraysPtr.ro_w[i+(j+1)*localNx+k*localNx*(*gpu_def).Ny] * (*gpu_def).g_const * (j+1) * (*gpu_def).hy)
				    - (P_w - DevArraysPtr.ro_w[i+j*localNx+k*localNx*(*gpu_def).Ny] * (*gpu_def).g_const * j * (*gpu_def).hy)) / (*gpu_def).hy;

				DevArraysPtr.uy_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * (
					(DevArraysPtr.P_n[i+(j+1)*localNx+k*localNx*(*gpu_def).Ny] - DevArraysPtr.ro_n[i+(j+1)*localNx+k*localNx*(*gpu_def).Ny] * (*gpu_def).g_const * (j+1) * (*gpu_def).hy)
					- (P_n - DevArraysPtr.ro_n[i+j*localNx+k*localNx*(*gpu_def).Ny] * (*gpu_def).g_const * j * (*gpu_def).hy)) / (*gpu_def).hy;			}

			if (j == ((*gpu_def).Ny) - 1)
			{
				DevArraysPtr.uy_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * (
					(P_w - DevArraysPtr.ro_w[i+j*localNx+k*localNx*(*gpu_def).Ny] * (*gpu_def).g_const * j * (*gpu_def).hy)
					- (DevArraysPtr.P_w[i+(j-1)*localNx+k*localNx*(*gpu_def).Ny] - DevArraysPtr.ro_w[i+(j-1)*localNx+k*localNx*(*gpu_def).Ny] * (*gpu_def).g_const * (j-1) * (*gpu_def).hy)) / (*gpu_def).hy;

				DevArraysPtr.uy_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * (
					(P_n - DevArraysPtr.ro_n[i+j*localNx+k*localNx*(*gpu_def).Ny] * (*gpu_def).g_const * j * (*gpu_def).hy)
					- (DevArraysPtr.P_n[i+(j-1)*localNx+k*localNx*(*gpu_def).Ny] - DevArraysPtr.ro_n[i+(j-1)*localNx+k*localNx*(*gpu_def).Ny] * (*gpu_def).g_const * (j-1) * (*gpu_def).hy)) / (*gpu_def).hy;
			}
			if ((j != 0) && (j != ((*gpu_def).Ny) - 1))
			{
				DevArraysPtr.uy_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * (
					(DevArraysPtr.P_w[i+(j+1)*localNx+k*localNx*(*gpu_def).Ny] - DevArraysPtr.ro_w[i+(j+1)*localNx+k*localNx*(*gpu_def).Ny] * (*gpu_def).g_const * (j+1) * (*gpu_def).hy)
					- (DevArraysPtr.P_w[i+(j-1)*localNx+k*localNx*(*gpu_def).Ny] - DevArraysPtr.ro_w[i+(j-1)*localNx+k*localNx*(*gpu_def).Ny] * (*gpu_def).g_const * (j-1) * (*gpu_def).hy)) / (2*(*gpu_def).hy);

				DevArraysPtr.uy_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * (
					(DevArraysPtr.P_n[i+(j+1)*localNx+k*localNx*(*gpu_def).Ny] - DevArraysPtr.ro_n[i+(j+1)*localNx+k*localNx*(*gpu_def).Ny] * (*gpu_def).g_const * (j+1) * (*gpu_def).hy)
					- (DevArraysPtr.P_n[i+(j-1)*localNx+k*localNx*(*gpu_def).Ny] - DevArraysPtr.ro_n[i+(j-1)*localNx+k*localNx*(*gpu_def).Ny] * (*gpu_def).g_const * (j-1) * (*gpu_def).hy)) / (2*(*gpu_def).hy);
			}
		}
		else
		{
			DevArraysPtr.uy_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
			DevArraysPtr.uy_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
		}

		if (((*gpu_def).Nz)>2)
		{
			if (k == 0)
			{
				DevArraysPtr.uz_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * (DevArraysPtr.P_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - P_w) / ((*gpu_def).hz);
				DevArraysPtr.uz_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * (DevArraysPtr.P_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - P_n) / ((*gpu_def).hz);
			}
			if (k == ((*gpu_def).Nz) - 1)
			{
				DevArraysPtr.uz_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * (P_w - DevArraysPtr.P_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / ((*gpu_def).hz);
				DevArraysPtr.uz_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * (P_n - DevArraysPtr.P_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / ((*gpu_def).hz);
			}
			if ((k != 0) && (k != ((*gpu_def).Nz) - 1))
			{
				DevArraysPtr.uz_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * ((DevArraysPtr.P_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - DevArraysPtr.P_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hz)));
				DevArraysPtr.uz_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * ((DevArraysPtr.P_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - DevArraysPtr.P_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hz)));
			}
		}
		else
		{
			DevArraysPtr.uz_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
			DevArraysPtr.uz_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
		}

	device_test_nan(DevArraysPtr.ux_w[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
	device_test_nan(DevArraysPtr.ux_n[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
	device_test_nan(DevArraysPtr.uy_w[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
	device_test_nan(DevArraysPtr.uy_n[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
	device_test_nan(DevArraysPtr.uz_w[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
	device_test_nan(DevArraysPtr.uz_n[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
	}
}

// ������ ��������� �� ���� ������ �����
void u_calculation(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ, consts def)
{
	assign_u_kernel<<<dim3(blocksX,blocksY,blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,rank,size); 
	checkErrors("assign u");
	cudaPrintfDisplay(stdout, true);
}

// ������ ro*S � ������ ����� ����� ������� ������������ ���������
__global__ void assign_roS_kernel_nr(ptr_Arrays DevArraysPtr, int localNx, double t)
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int j=threadIdx.y+blockIdx.y*blockDim.y;
	int k=threadIdx.z+blockIdx.z*blockDim.z;
	
	if ((i<localNx-1) && (j<(*gpu_def).Ny-1) && (k<(*gpu_def).Nz) && (i!=0) && (j!=0) && (((k!=0) && (k!=(*gpu_def).Nz-1)) || ((*gpu_def).Nz<2)))
	{
		int media = DevArraysPtr.media[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		double S2 = DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double roS1 = DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - S2);
		double roS2 = DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * S2;
 		double P1 = DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P2 = DevArraysPtr.P_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		double x1, x2, y1, y2, z1, z2, f1, f2, f3, g1, g2, g3;

		if (((*gpu_def).Nz)<2)
		{
			f3=0;
			g3=0;
		}
		else
		{
			z2 = -(DevArraysPtr.P_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - P1)/(*gpu_def).hz;
			z1 = -(P1 - DevArraysPtr.P_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)])/(*gpu_def).hz;

			f3 = (((z2 + abs(z2))/2.0 - (z1 - abs(z1))/2.0)*(-1) * DevArraysPtr.Xi_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                      (z1 + abs(z1))/2.0*(-1)* DevArraysPtr.Xi_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] +
                      (z2 - abs(z2))/2.0*(-1)* DevArraysPtr.Xi_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)])/(*gpu_def).hz;

			z2 = -(DevArraysPtr.P_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - P2)/(*gpu_def).hz;
			z1 = -(P2 - DevArraysPtr.P_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)])/(*gpu_def).hz;

			g3 = (((z2 + abs(z2))/2.0 - (z1 - abs(z1))/2.0)*(-1) * DevArraysPtr.Xi_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                      (z1 + abs(z1))/2.0*(-1)* DevArraysPtr.Xi_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] +
                      (z2 - abs(z2))/2.0*(-1)* DevArraysPtr.Xi_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)])/(*gpu_def).hz;
		}

		x2 = -(DevArraysPtr.P_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - P1)/(*gpu_def).hx;
        x1 = -(P1 - DevArraysPtr.P_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hx;

        y2 = -(DevArraysPtr.P_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - P1)/(*gpu_def).hy + DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (*gpu_def).g_const;
        y1 = -(P1 - DevArraysPtr.P_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hy + DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (*gpu_def).g_const;

        f1 = (((x2 + abs(x2))/2.0 - (x1 - abs(x1))/2.0)*(-1) * DevArraysPtr.Xi_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                (x1 + abs(x1))/2.0*(-1)* DevArraysPtr.Xi_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] +
                (x2 - abs(x2))/2.0*(-1)* DevArraysPtr.Xi_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hx;

        f2 = (((y2 + abs(y2))/2.0 - (y1 - abs(y1))/2.0)*(-1)* DevArraysPtr.Xi_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                (y1 + abs(y1))/2.0*(-1)* DevArraysPtr.Xi_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] +
                (y2 - abs(y2))/2.0*(-1)* DevArraysPtr.Xi_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hy;


        x2 = -(DevArraysPtr.P_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - P2)/(*gpu_def).hx;
        x1 = -(P2 - DevArraysPtr.P_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hx;

        y2 = -(DevArraysPtr.P_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - P2)/(*gpu_def).hy + DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (*gpu_def).g_const;
        y1 = -(P2 - DevArraysPtr.P_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hy + DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (*gpu_def).g_const;

        g1 = (((x2 + abs(x2))/2.0 - (x1 - abs(x1))/2.0)*(-1) * DevArraysPtr.Xi_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                (x1 + abs(x1))/2.0*(-1)* DevArraysPtr.Xi_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] +
                (x2 - abs(x2))/2.0*(-1)* DevArraysPtr.Xi_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hx;

        g2 = (((y2 + abs(y2))/2.0 - (y1 - abs(y1))/2.0)*(-1)* DevArraysPtr.Xi_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                (y1 + abs(y1))/2.0*(-1)* DevArraysPtr.Xi_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] +
                (y2 - abs(y2))/2.0*(-1)* DevArraysPtr.Xi_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hy;

		DevArraysPtr.roS_w_old[i+j*localNx+k*localNx*((*gpu_def).Ny)] = roS1;
		DevArraysPtr.roS_n_old[i+j*localNx+k*localNx*((*gpu_def).Ny)] = roS2;
		DevArraysPtr.roS_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = roS1 - ((*gpu_def).dt/(*gpu_def).m[media])*(f1 + f2 + f3);
		DevArraysPtr.roS_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = roS2 - ((*gpu_def).dt/(*gpu_def).m[media])*(g1 + g2 + g3);

		device_test_positive(DevArraysPtr.roS_w_old[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr.roS_n_old[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr.roS_w[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr.roS_n[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
	}
}

// ������ ro*S � ������ ����� �����
__global__ void assign_roS_kernel(ptr_Arrays DevArraysPtr, int localNx, double t) 
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int j=threadIdx.y+blockIdx.y*blockDim.y;
	int k=threadIdx.z+blockIdx.z*blockDim.z;

	if ((i<localNx-1) && (j<(*gpu_def).Ny-1) && (k<(*gpu_def).Nz) && (i!=0) && (j!=0) && (((k!=0) && (k!=(*gpu_def).Nz-1)) || ((*gpu_def).Nz<2)))
	{
		int local=i+j*localNx+k*localNx*((*gpu_def).Ny);
		int media = DevArraysPtr.media[local];
		double S_n = DevArraysPtr.S_n[local];
		double roS_w = DevArraysPtr.ro_w[local] * (1 - S_n);
		double roS_n = DevArraysPtr.ro_n[local] * S_n;

		double divgrad1, divgrad2, Tx1, Ty1, Tx2, Ty2, Tz1, Tz2, A1=0, A2=0;

		if (((*gpu_def).Nz)<2)
		{
			divgrad1=0;
			divgrad2=0;
			Tz1=0;
			Tz2=0;
		}
		else
		{
			divgrad1 = ((*gpu_def).m[media] * ((*gpu_def).l) * ((*gpu_def).c_w) / 2.) * (DevArraysPtr.ro_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * (1. - DevArraysPtr.S_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)]) - 2 * DevArraysPtr.ro_w[local] * (1. - S_n) + DevArraysPtr.ro_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * (1. - DevArraysPtr.S_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)])) / (((*gpu_def).hz) * ((*gpu_def).hz));
			divgrad2 = ((*gpu_def).m[media] * ((*gpu_def).l) * ((*gpu_def).c_n) / 2.) * (DevArraysPtr.ro_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.S_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - 2 * DevArraysPtr.ro_n[local] * S_n + DevArraysPtr.ro_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * (DevArraysPtr.S_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)])) / (((*gpu_def).hz) * ((*gpu_def).hz));
			Tz1 = (DevArraysPtr.ro_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.uz_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.uz_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / (2. * ((*gpu_def).hz));
			Tz2 = (DevArraysPtr.ro_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.uz_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.uz_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / (2. * ((*gpu_def).hz));
		}

		divgrad1 += ((*gpu_def).m[media] * ((*gpu_def).l) * ((*gpu_def).c_w) / 2.) *
		((DevArraysPtr.ro_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - DevArraysPtr.S_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)]) - 2 * DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - S_n) + DevArraysPtr.ro_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - DevArraysPtr.S_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)])) / (((*gpu_def).hx) * ((*gpu_def).hx)) +
		(DevArraysPtr.ro_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * (1 - DevArraysPtr.S_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)]) - 2 * DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - S_n) + DevArraysPtr.ro_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * (1 - DevArraysPtr.S_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)])) / (((*gpu_def).hy) * ((*gpu_def).hy)));

		divgrad2 += ((*gpu_def).m[media] * ((*gpu_def).l) * ((*gpu_def).c_n) / 2.) *
		((DevArraysPtr.ro_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.S_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - 2 * DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * S_n + DevArraysPtr.ro_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.S_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (((*gpu_def).hx) * ((*gpu_def).hx)) +
		(DevArraysPtr.ro_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.S_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - 2 * DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * S_n + DevArraysPtr.ro_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.S_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / (((*gpu_def).hy) * ((*gpu_def).hy)));

		Tx1 = (DevArraysPtr.ro_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ux_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ux_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hx));
		Ty1 = (DevArraysPtr.ro_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.uy_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.uy_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hy));
		Tx2 = (DevArraysPtr.ro_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ux_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ux_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hx));
		Ty2 = (DevArraysPtr.ro_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.uy_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.uy_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hy));

		if ((t < 2 * ((*gpu_def).dt)) || TWO_LAYERS)
		{
			A1 = roS_w + (((*gpu_def).dt) / (*gpu_def).m[media]) * (divgrad1 - Tx1 - Ty1 - Tz1);
			A2 = roS_n + (((*gpu_def).dt) / (*gpu_def).m[media]) * (divgrad2 - Tx2 - Ty2 - Tz2);
		}
		else
		{
			A1 = (2 * ((*gpu_def).dt) * ((*gpu_def).dt)) / ((*gpu_def).m[media] * (((*gpu_def).dt) + 2 * ((*gpu_def).tau))) * (divgrad1 - Tx1 - Ty1 - Tz1 + (2 * roS_w * (*gpu_def).m[media] * ((*gpu_def).tau)) / (((*gpu_def).dt) * ((*gpu_def).dt)) + DevArraysPtr.roS_w_old[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (*gpu_def).m[media] * (((*gpu_def).dt) - 2 * ((*gpu_def).tau)) / (2 * ((*gpu_def).dt) * ((*gpu_def).dt)));
			A2 = (2 * ((*gpu_def).dt) * ((*gpu_def).dt)) / ((*gpu_def).m[media] * (((*gpu_def).dt) + 2 * ((*gpu_def).tau))) * (divgrad2 - Tx2 - Ty2 - Tz2 + (2 * roS_n * (*gpu_def).m[media] * ((*gpu_def).tau)) / (((*gpu_def).dt) * ((*gpu_def).dt)) + DevArraysPtr.roS_n_old[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (*gpu_def).m[media] * (((*gpu_def).dt) - 2 * ((*gpu_def).tau)) / (2 * ((*gpu_def).dt) * ((*gpu_def).dt)));
		}

		DevArraysPtr.roS_w_old[local] = roS_w;
		DevArraysPtr.roS_n_old[local] = roS_n;
		DevArraysPtr.roS_w[local] = A1;
		DevArraysPtr.roS_n[local] = A2;

		device_test_positive(DevArraysPtr.roS_w_old[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr.roS_n_old[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr.roS_w[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
		device_test_positive(DevArraysPtr.roS_n[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
	}
}

// ������ ro*S �� ���� ������ �����
void roS_calculation(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, consts def, double t, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ)
{
	#ifdef NR
		assign_roS_kernel_nr<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,t); 
	#else
		assign_roS_kernel<<<dim3(blocksX,blocksY,blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,t);
	#endif
		checkErrors("assign roS");
		cudaPrintfDisplay(stdout, true);
}

// ��������� ������� �� S2
__global__ void Sn_boundary_kernel(ptr_Arrays DevArraysPtr, int localNx, int rank, int size) 
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int j=threadIdx.y+blockIdx.y*blockDim.y;
	int k=threadIdx.z+blockIdx.z*blockDim.z;

	if ((i<localNx) && (j<((*gpu_def).Ny)) && (k<((*gpu_def).Nz)) && (device_is_active_point(i, localNx, rank, size)==1))
	{
		if ((i == 0) && ((*gpu_def).Nx>2) && (j>0) && (j<(*gpu_def).Ny - 1))
		{
		   DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)];
		   return;
		}

		if ((i == localNx - 1) && (((*gpu_def).Nx)>2) && (j>0) && (j<(*gpu_def).Ny - 1))
		{
			DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)];
			 return;
		}

		if ((j == ((*gpu_def).Ny) - 1) && (((*gpu_def).Ny)>2))
		{
			DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)];

			if (i==0) 
				DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)];
			if (i==(*gpu_def).Nx - 1)
				DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)];
			return;
		}
		
		if ((j==0) && (((*gpu_def).Ny)>2))
		{
			int I=device_i_to_I(i,rank,size, gpu_def[0]);
			if ((I>=((*gpu_def).Nx)/2-((*gpu_def).source)) && (I<=((*gpu_def).Nx)/2+((*gpu_def).source)) && (k>=((*gpu_def).Nz)/2-((*gpu_def).source)) && (k<=((*gpu_def).Nz)/2+((*gpu_def).source)))
				DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = (*gpu_def).S_n_gr;
			else
				DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)];
			return;
		}

		if ((k == 0) && ((*gpu_def).Nz > 2) && (j>0) && (j<(*gpu_def).Ny - 1))
		{
			DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)];
			return;
		}
		
		if ((k == (*gpu_def).Nz - 1) && ((*gpu_def).Nz > 2) && (j>0) && (j<(*gpu_def).Ny - 1))
		{
			DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)];
			return;
		}

		device_test_positive(DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
	}
}

// ��������� ������� �� P1
__global__ void Pw_boundary_kernel(ptr_Arrays DevArraysPtr, int localNx, int rank, int size) 
{
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int j=threadIdx.y+blockIdx.y*blockDim.y;
	int k=threadIdx.z+blockIdx.z*blockDim.z;

	if ((i<localNx) && (j<((*gpu_def).Ny)) && (k<((*gpu_def).Nz)) && (device_is_active_point(i, localNx, rank, size)==1))
	{
		if ((i == 0) && ((*gpu_def).Nx > 2) && (j>0) && (j<(*gpu_def).Ny - 1))
		{
			DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)]; 
			//return;
		}

		if ((i == localNx - 1) && ((*gpu_def).Nx>2) && (j>0) && (j<(*gpu_def).Ny - 1))
		{
			DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)];
			//return;
		}

		if ((j == ((*gpu_def).Ny) - 1) && (((*gpu_def).Ny)>2))
		{
			DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P_w[i+(j-1)*localNx+k*localNx*(*gpu_def).Ny] + DevArraysPtr.ro_w[i+(j-1)*localNx+k*localNx*(*gpu_def).Ny] * (*gpu_def).g_const * (*gpu_def).hy;
			if (i==0) 
				DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)];
			if (i==(*gpu_def).Nx - 1)
				DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)];
			//return;
		}

		if ((j==0) && (((*gpu_def).Ny)>2))
		{
			DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = (*gpu_def).P_atm;
			//return;
		}

		if ((k == 0) && (((*gpu_def).Nz)>2) && (j>0) && (j<(*gpu_def).Ny - 1))
		{
			DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)]; 
			//return;
		}

		if ((k == ((*gpu_def).Nz) - 1) && (((*gpu_def).Nz)>2) && (j>0) && (j<(*gpu_def).Ny - 1))
		{
			DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)];
			//return;
		}

		device_test_positive(DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)], __FILE__, __LINE__);
	}
}

// ���������� ��������� �������
void boundary_conditions(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ, consts def)
{
	Sn_boundary_kernel<<<dim3(blocksX,blocksY,blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,rank,size); 
	checkErrors("assign Sn");
	cudaPrintfDisplay(stdout, true);

	Pw_boundary_kernel<<<dim3(blocksX,blocksY,blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,rank,size); 
	checkErrors("assign Pw");
	cudaPrintfDisplay(stdout, true);
}	

// ������� �������� ������ � ������ �����
void load_data_to_host(double* HostArrayPtr, double* DevArrayPtr, int localNx, consts def)
{
	hipMemcpy( HostArrayPtr, DevArrayPtr, localNx*(def.Ny)*(def.Nz)*sizeof(double), hipMemcpyDeviceToHost );
	checkErrors("copy data to host");
}

// ������� �������� ������ ���� double � ������ ����������
void load_data_to_device(double* HostArrayPtr, double* DevArrayPtr, int localNx, consts def)
{
	hipMemcpy( DevArrayPtr, HostArrayPtr, localNx*(def.Ny)*(def.Nz)*sizeof(double), hipMemcpyHostToDevice );
	checkErrors("copy double data to device");
}

// ������� �������� ������ ���� int � ������ ����������
void load_data_to_device_int(int* HostArrayPtr, int* DevArrayPtr, int localNx, consts def)
{
	hipMemcpy( DevArrayPtr, HostArrayPtr, localNx*(def.Ny)*(def.Nz)*sizeof(int), hipMemcpyHostToDevice );
	checkErrors("copy int data to device");
}

// ��������� ������ ���������� ��� ������ ����� ��������� �������
void device_memory_allocation(ptr_Arrays* ArraysPtr, double** DevBuffer, int localNx, consts def)
{
	hipMalloc((void**) DevBuffer,  2 * (def.Ny) * (def.Nz) * sizeof(double));

	hipMalloc((void**) &((*ArraysPtr).x),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).y),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).z),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).P_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).P_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).S_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).ro_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).ro_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).ux_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).uy_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).uz_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).ux_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).uy_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).uz_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).Xi_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).Xi_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).roS_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).roS_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).roS_w_old),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).roS_n_old),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).media),  localNx * (def.Ny) * (def.Nz) * sizeof(int));

	checkErrors("memory allocation");
}

// ����������� ������ ���������� �� ��� ������� ����� ��������� �������
void device_memory_free(ptr_Arrays DevArraysPtr, double* DevBuffer)
{
	hipFree (DevBuffer);
	hipFree (DevArraysPtr.x);
	hipFree (DevArraysPtr.y);
	hipFree (DevArraysPtr.z);
	hipFree (DevArraysPtr.P_w);
	hipFree (DevArraysPtr.P_n);
	hipFree (DevArraysPtr.S_n);
	hipFree (DevArraysPtr.ro_w);
	hipFree (DevArraysPtr.ro_n);
	hipFree (DevArraysPtr.ux_w);
	hipFree (DevArraysPtr.uy_w);
	hipFree (DevArraysPtr.uz_w);
	hipFree (DevArraysPtr.ux_n);
	hipFree (DevArraysPtr.uy_n);
	hipFree (DevArraysPtr.uz_n);
	hipFree (DevArraysPtr.Xi_w);
	hipFree (DevArraysPtr.Xi_n);
	hipFree (DevArraysPtr.roS_w);
	hipFree (DevArraysPtr.roS_n);
	hipFree (DevArraysPtr.roS_w_old);
	hipFree (DevArraysPtr.roS_n_old);
	hipFree (DevArraysPtr.media);

	checkErrors("memory reliase");
}

// ������������� ����������
// ������ ���������� �� ����������, ����� �������� �����
// ������ ������������ ����������
void device_initialization(int rank, int* blocksX, int* blocksY, int* blocksZ, int localNx, consts def)
{
	// ���� 3 ���������� �� ����� ���� � ������� ����������� ����
	int device=rank%3;
	hipSetDevice(device);
	printf("CUDA initialized.\n");

	// ���������� ����������� ������
	// ���� ����� ����� ����� �� ������ ������� �����,
	// �� ���������� ������ ����� �� 1 ������.
	*blocksX=localNx/BlockNX;
	if (localNx%BlockNX!=0)
		(*blocksX)++;
	*blocksY=(def.Ny)/BlockNY;
	if ((def.Ny)%BlockNY!=0)
		(*blocksY)++;
	*blocksZ=(def.Nz)/BlockNZ;
	if ((def.Nz)%BlockNZ!=0)
		(*blocksZ)++;

	consts* deff=new consts[1];
	deff[0]=def;

	hipMemcpyToSymbol(HIP_SYMBOL( gpu_def), deff, sizeof ( consts ));//, 0, hipMemcpyHostToDevice );
	checkErrors("constant memory copy");

	int deviceCount;
    hipDeviceProp_t devProp;
    hipGetDeviceCount ( &deviceCount );

        hipGetDeviceProperties ( &devProp, device );
        printf ( "Device %d\n", device );
        printf ( "Compute capability : %d.%d\n", devProp.major, devProp.minor );
        printf ( "Name : %s\n", devProp.name );
        printf ( "Total Global Memory : %ld\n", devProp.totalGlobalMem );
        printf ( "Shared memory per block: %d\n", devProp.sharedMemPerBlock );
        printf ( "Registers per block : %d\n", devProp.regsPerBlock );
        printf ( "Warp size : %d\n", devProp.warpSize );
        printf ( "Max threads per block : %d\n", devProp.maxThreadsPerBlock );
        printf ( "Total constant memory : %d\n", devProp.totalConstMem );
		printf ( "Number of multiprocessors: %d\n",  devProp.multiProcessorCount);
		printf ( "Kernel execution timeout: %s\n\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
		for (int i = 0; i < 3; ++i)
			printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
		for (int i = 0; i < 3; ++i)
			printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);


		// ������������ ������ ��������� ����� ��� ����������
		// 21 - ���������� ���������� � �����
		printf ( "\nTotal NAPL_Filtration grid size : %d\n\n", devProp.totalGlobalMem/(21*sizeof(double)) );

		// localNX+2 ������ ��� 2NyNz �� ����� ������ ����������
		if ((localNx+2)*(def.Ny)*(def.Nz) > (devProp.totalGlobalMem/(21*sizeof(double))))
			printf ("\nError! Not enough memory at GPU, rank=%d\n",rank);
		fflush( stdout);

		// �������������� ���������� cuPrintf ��� ������ ������ �� �������
		// ����� �� kernel
		cudaPrintfInit();
}

// ����������� ����������
void device__finalization(void)
{
	// ������������� ���������� cuPrintf ��� ������ ������ �� �������
	// ����� �� kernel
	cudaPrintfEnd();
}

__global__ void load_exchange_data_kernel(double* DevArrayPtr, double* DevBuffer, int localNx)
{
	int j=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.y+blockIdx.y*blockDim.y;

	if (j<(*gpu_def).Ny && k<(*gpu_def).Nz)
	{
		DevBuffer[j+((*gpu_def).Ny)*k]=DevArrayPtr[1+localNx*j+localNx*((*gpu_def).Ny)*k];
		DevBuffer[j+((*gpu_def).Ny)*k+((*gpu_def).Ny)*((*gpu_def).Nz)]=DevArrayPtr[localNx-2+localNx*j+localNx*((*gpu_def).Ny)*k];

		device_test_nan(DevBuffer[j+((*gpu_def).Ny)*k], __FILE__, __LINE__);
		device_test_nan(DevBuffer[j+((*gpu_def).Ny)*k+((*gpu_def).Ny)*((*gpu_def).Nz)], __FILE__, __LINE__);
	}
}


void load_exchange_data(double* HostArrayPtr, double* DevArrayPtr, double* HostBuffer, double* DevBuffer, int localNx, int blocksY, int blocksZ, int rank, int size, consts def)
{
	load_exchange_data_kernel<<<dim3(blocksY,blocksZ), dim3(BlockNY,BlockNZ)>>>(DevArrayPtr, DevBuffer, localNx); 
	checkErrors("load_exchange_data");
	cudaPrintfDisplay(stdout, true);

	hipMemcpy( HostBuffer, DevBuffer, 2*(def.Ny)*(def.Nz)*sizeof(double), hipMemcpyDeviceToHost );
	checkErrors("copy data to host");
	cudaPrintfDisplay(stdout, true);
}

__global__ void save_exchange_data_kernel(double* DevArrayPtr, double* DevBuffer, int localNx, int rank, int size)
{
	int j=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.y+blockIdx.y*blockDim.y;

	if (j<(*gpu_def).Ny && k<(*gpu_def).Nz)
	{
		if (rank!=0)
		{
			DevArrayPtr[localNx*j+localNx*((*gpu_def).Ny)*k]=DevBuffer[j+((*gpu_def).Ny)*k];
			device_test_nan(DevArrayPtr[localNx*j+localNx*((*gpu_def).Ny)*k], __FILE__, __LINE__);
		}
		if (rank!=size-1)
		{
			DevArrayPtr[localNx-1+localNx*j+localNx*((*gpu_def).Ny)*k]=DevBuffer[j+((*gpu_def).Ny)*k+((*gpu_def).Ny)*((*gpu_def).Nz)];
			device_test_nan(DevArrayPtr[localNx-1+localNx*j+localNx*((*gpu_def).Ny)*k], __FILE__, __LINE__);
		}
	}
}

void save_exchange_data(double* HostArrayPtr, double* DevArrayPtr, double* HostBuffer, double* DevBuffer, int localNx, int blocksY, int blocksZ, int rank, int size, consts def)
{
	hipMemcpy( DevBuffer, HostBuffer, 2*(def.Ny)*(def.Nz)*sizeof(double), hipMemcpyHostToDevice );
	checkErrors("copy data to device");
	cudaPrintfDisplay(stdout, true);

	save_exchange_data_kernel<<<dim3(blocksY,blocksZ), dim3(BlockNY,BlockNZ)>>>(DevArrayPtr, DevBuffer, localNx, rank, size); 
	checkErrors("save_exchange_data");
	cudaPrintfDisplay(stdout, true);
}