#include "hip/hip_runtime.h"
#include "defines.h"
#include <hip/hip_runtime.h>

__constant__ consts gpu_def [1];

// �������� ������ GPU
void checkErrors(char *label) 
{
#ifdef TEST
	hipError_t err;

	err = hipDeviceSynchronize();
	if (err != hipSuccess)
	{
		char *e = (char*) hipGetErrorString(err);
		printf("CUDA Error: %s (at %s)\n", e, label);
	}

	err=hipGetLastError();
	if (err != hipSuccess)
	{
		char *e = (char*) hipGetErrorString(err);
		printf("CUDA Error: %s (at %s)\n", e, label);
		fflush(stdout);
	}
#endif
}

// �������������� ��������� ��������� ���������� � ����������
// ������ ��������� �������� �������������� ����� � ������� ���
// ������ �������, ���� ����� ������ 
// (���� 2 ������ � ����� ������,�� +2 �����). 
// ���������� ������� �������� ��� ������� ����� (������ � ������� �� rank==0)
__device__ int device_i_to_I(int i, int rank, int size, consts def)
{
	int I;
	if (rank <= ((*gpu_def).Nx)%size)
	{
		if(rank==0)
			I=i;
		else
			I=(((*gpu_def).Nx)/size+1)*rank+i-1;
	}
	else
		I=(((*gpu_def).Nx)/size+1)*rank-(rank-((*gpu_def).Nx)%size)+i-1;
	return I;
}

// �������� �� ����� �������� (�.�. �� ��������������� ������ ��� ������ �� ��������)
__device__ int device_is_active_point(int i, int localNx, int rank, int size)
{
	if((rank!=0 && i==0) || (rank!=size-1 && i==localNx-1))
		return 0;
	else
		return 1;
}

// ������ ����������, �������� NAPL P2 � Xi � ������ ����� ����� (���������� �� ��������� �����)
__global__ void assign_ro_Pn_Xi_kernel(ptr_Arrays DevArraysPtr, int localNx, int rank, int size) 
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;

	if ((i<localNx) && (j<((*gpu_def).Ny)) && (k<((*gpu_def).Nz)) && (device_is_active_point(i, localNx, rank, size)==1))
	{
		int media = DevArraysPtr.media[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double S_n = DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P_w = DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		double S_e = (1 - S_n - (*gpu_def).S_wr[media]) / (1 - (*gpu_def).S_wr[media]);
		double k_w = pow(S_e, (2. + 3. * (*gpu_def).lambda[media]) / (*gpu_def).lambda[media]);
		double k_n = (1. - S_e) * (1. - S_e) * (1 - pow(S_e, (2. + (*gpu_def).lambda[media]) / (*gpu_def).lambda[media]));
		double P_k = (*gpu_def).P_d[media] * pow((1 - S_n - (*gpu_def).S_wr[media]) / (1 - (*gpu_def).S_wr[media]), -1 / (*gpu_def).lambda[media]);

		DevArraysPtr.P_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = P_w + P_k;
		DevArraysPtr.Xi_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = -1 * (*gpu_def).K[media] * k_w / mu_w;
		DevArraysPtr.Xi_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = -1 * (*gpu_def).K[media] * k_n / mu_n;
		DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = ro0_w * (1 + ((*gpu_def).beta_w) * (P_w - P_atm));
		DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = ro0_n * (1 + ((*gpu_def).beta_n) * (P_w + P_k - P_atm));
	}
}

// ������ ����������, �������� NAPL P2 � Xi �� ���� ������ �����
void ro_Pn_Xi_calculation(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, consts def, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ)
{
	assign_ro_Pn_Xi_kernel<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,rank,size); 
	checkErrors("assign Pn, Xi, ro");
}

// ������ �������� � ������ ����� �����
__global__ void assign_u_kernel(ptr_Arrays DevArraysPtr, int localNx, int rank, int size) 
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;

	if ((i<localNx) && (j<((*gpu_def).Ny)) && (k<((*gpu_def).Nz)) && (device_is_active_point(i, localNx, rank, size)==1))
	{
		double Xi_w = DevArraysPtr.Xi_w[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double Xi_n = DevArraysPtr.Xi_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P_w = DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P_n = DevArraysPtr.P_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		if (((*gpu_def).Nx)>2)
		{
			if (i == 0)
			{
				DevArraysPtr.ux_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * (DevArraysPtr.P_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - P_w) / ((*gpu_def).hx);
				DevArraysPtr.ux_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * (DevArraysPtr.P_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - P_n) / ((*gpu_def).hx);
			}
			if (i == localNx - 1)
			{
				DevArraysPtr.ux_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * (P_w - DevArraysPtr.P_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / ((*gpu_def).hx);
				DevArraysPtr.ux_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * (P_n - DevArraysPtr.P_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / ((*gpu_def).hx);
			}
			if ((i != 0) && (i != localNx - 1))
			{
				DevArraysPtr.ux_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * ((DevArraysPtr.P_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.P_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hx)));
				DevArraysPtr.ux_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * ((DevArraysPtr.P_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.P_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hx)));
			}
		}
		else
		{
			DevArraysPtr.ux_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
			DevArraysPtr.ux_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
		}
	
		if (((*gpu_def).Ny)>2)
		{
			if (j == 0)
			{
				DevArraysPtr.uy_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * ((DevArraysPtr.P_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - P_w) / ((*gpu_def).hy) - DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const);
				DevArraysPtr.uy_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * ((DevArraysPtr.P_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - P_n) / ((*gpu_def).hy) - DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const);
			}

			if (j == ((*gpu_def).Ny) - 1)
			{
				DevArraysPtr.uy_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * ((P_w - DevArraysPtr.P_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / ((*gpu_def).hy) - DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const);
				DevArraysPtr.uy_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * ((P_n - DevArraysPtr.P_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / ((*gpu_def).hy) - DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const);
			}
			if ((j != 0) && (j != ((*gpu_def).Ny) - 1))
			{
				DevArraysPtr.uy_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * ((DevArraysPtr.P_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.P_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hy)) - DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const);
				DevArraysPtr.uy_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * ((DevArraysPtr.P_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.P_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hy)) - DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const);
			}
		}
		else
		{
			DevArraysPtr.uy_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
			DevArraysPtr.uy_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
		}

		if (((*gpu_def).Nz)>2)
		{
			if (k == 0)
			{
				DevArraysPtr.uz_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * (DevArraysPtr.P_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - P_w) / ((*gpu_def).hz);
				DevArraysPtr.uz_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * (DevArraysPtr.P_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - P_n) / ((*gpu_def).hz);
			}
			if (k == ((*gpu_def).Nz) - 1)
			{
				DevArraysPtr.uz_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * (P_w - DevArraysPtr.P_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / ((*gpu_def).hz);
				DevArraysPtr.uz_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * (P_n - DevArraysPtr.P_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / ((*gpu_def).hz);
			}
			if ((k != 0) && (i != ((*gpu_def).Nz) - 1))
			{
				DevArraysPtr.uz_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_w * ((DevArraysPtr.P_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - DevArraysPtr.P_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hz)));
				DevArraysPtr.uz_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = Xi_n * ((DevArraysPtr.P_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - DevArraysPtr.P_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hz)));
			}
		}
		else
		{
			DevArraysPtr.uz_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
			DevArraysPtr.uz_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
		}
	}
}

// ������ ��������� �� ���� ������ �����
void u_calculation(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ, consts def)
{
	assign_u_kernel<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,rank,size); 
	checkErrors("assign u");
}
// ������ ro*S � ������ ����� ����� ������� ������������ ���������
__global__ void assign_roS_kernel_nr(ptr_Arrays DevArraysPtr, int localNx, double t)
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;
	
	if ((i<localNx-1) && (j<(*gpu_def).Ny-1) && (k<(*gpu_def).Nz) && (i!=0) && (j!=0) && (((k!=0) && (k!=(*gpu_def).Nz-1)) || ((*gpu_def).Nz<2)))
	{
		int media = DevArraysPtr.media[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		double S2 = DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double roS1 = DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - S2);
		double roS2 = DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * S2;
 		double P1 = DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P2 = DevArraysPtr.P_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];

		double x1, x2, y1, y2, z1, z2, f1, f2, f3, g1, g2, g3;

		if (((*gpu_def).Nz)<2)
		{
			f3=0;
			g3=0;
		}
		else
		{
			z2 = -(DevArraysPtr.P_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - P1)/(*gpu_def).hz;
			z1 = -(P1 - DevArraysPtr.P_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)])/(*gpu_def).hz;

			f3 = (((z2 + abs(z2))/2.0 - (z1 - abs(z1))/2.0)*(-1) * DevArraysPtr.Xi_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                      (z1 + abs(z1))/2.0*(-1)* DevArraysPtr.Xi_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] +
                      (z2 - abs(z2))/2.0*(-1)* DevArraysPtr.Xi_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)])/(*gpu_def).hz;

			z2 = -(DevArraysPtr.P_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - P2)/(*gpu_def).hz;
			z1 = -(P2 - DevArraysPtr.P_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)])/(*gpu_def).hz;

			g3 = (((z2 + abs(z2))/2.0 - (z1 - abs(z1))/2.0)*(-1) * DevArraysPtr.Xi_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                      (z1 + abs(z1))/2.0*(-1)* DevArraysPtr.Xi_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] +
                      (z2 - abs(z2))/2.0*(-1)* DevArraysPtr.Xi_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)])/(*gpu_def).hz;
		}

		x2 = -(DevArraysPtr.P_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - P1)/(*gpu_def).hx;
        x1 = -(P1 - DevArraysPtr.P_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hx;

        y2 = -(DevArraysPtr.P_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - P1)/(*gpu_def).hy + DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const;
        y1 = -(P1 - DevArraysPtr.P_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hy + DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const;

        f1 = (((x2 + abs(x2))/2.0 - (x1 - abs(x1))/2.0)*(-1) * DevArraysPtr.Xi_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                (x1 + abs(x1))/2.0*(-1)* DevArraysPtr.Xi_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] +
                (x2 - abs(x2))/2.0*(-1)* DevArraysPtr.Xi_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hx;

        f2 = (((y2 + abs(y2))/2.0 - (y1 - abs(y1))/2.0)*(-1)* DevArraysPtr.Xi_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                (y1 + abs(y1))/2.0*(-1)* DevArraysPtr.Xi_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] +
                (y2 - abs(y2))/2.0*(-1)* DevArraysPtr.Xi_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hy;


        x2 = -(DevArraysPtr.P_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - P2)/(*gpu_def).hx;
        x1 = -(P2 - DevArraysPtr.P_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hx;

        y2 = -(DevArraysPtr.P_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - P2)/(*gpu_def).hy + DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const;
        y1 = -(P2 - DevArraysPtr.P_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hy + DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * g_const;

        g1 = (((x2 + abs(x2))/2.0 - (x1 - abs(x1))/2.0)*(-1) * DevArraysPtr.Xi_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                (x1 + abs(x1))/2.0*(-1)* DevArraysPtr.Xi_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] +
                (x2 - abs(x2))/2.0*(-1)* DevArraysPtr.Xi_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hx;

        g2 = (((y2 + abs(y2))/2.0 - (y1 - abs(y1))/2.0)*(-1)* DevArraysPtr.Xi_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] -
                (y1 + abs(y1))/2.0*(-1)* DevArraysPtr.Xi_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] +
                (y2 - abs(y2))/2.0*(-1)* DevArraysPtr.Xi_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ro_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)])/(*gpu_def).hy;

		DevArraysPtr.roS_w_old[i+j*localNx+k*localNx*((*gpu_def).Ny)] = roS1;
		DevArraysPtr.roS_n_old[i+j*localNx+k*localNx*((*gpu_def).Ny)] = roS2;
		DevArraysPtr.roS_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = roS1 - ((*gpu_def).dt/(*gpu_def).m[media])*(f1 + f2 + f3);
		DevArraysPtr.roS_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = roS2 - ((*gpu_def).dt/(*gpu_def).m[media])*(g1 + g2 + g3);
	}
}

// ������ ro*S � ������ ����� �����
__global__ void assign_roS_kernel(ptr_Arrays DevArraysPtr, int localNx, double t) 
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;

	if ((i<localNx-1) && (j<(*gpu_def).Ny-1) && (k<(*gpu_def).Nz) && (i!=0) && (j!=0) && (((k!=0) && (k!=(*gpu_def).Nz-1)) || ((*gpu_def).Nz<2)))
	{
		int local=i+j*localNx+k*localNx*((*gpu_def).Ny);
		int media = DevArraysPtr.media[local];
		double S_n = DevArraysPtr.S_n[local];
		double roS_w = DevArraysPtr.ro_w[local] * (1 - S_n);
		double roS_n = DevArraysPtr.ro_n[local] * S_n;

		double divgrad1, divgrad2, Tx1, Ty1, Tx2, Ty2, Tz1, Tz2, A1=0, A2=0;

		if (((*gpu_def).Nz)<2)
		{
			divgrad1=0;
			divgrad2=0;
			Tz1=0;
			Tz2=0;
		}
		else
		{
			divgrad1 = ((*gpu_def).m[media] * ((*gpu_def).l_w) * ((*gpu_def).c) / 2.) * (DevArraysPtr.ro_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * (1. - DevArraysPtr.S_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)]) - 2 * DevArraysPtr.ro_w[local] * (1. - S_n) + DevArraysPtr.ro_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * (1. - DevArraysPtr.S_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)])) / (((*gpu_def).hz) * ((*gpu_def).hz));
			divgrad2 = ((*gpu_def).m[media] * ((*gpu_def).l_n) * ((*gpu_def).c) / 2.) * (DevArraysPtr.ro_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.S_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - 2 * DevArraysPtr.ro_n[local] * S_n + DevArraysPtr.ro_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * (DevArraysPtr.S_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)])) / (((*gpu_def).hz) * ((*gpu_def).hz));
			Tz1 = (DevArraysPtr.ro_w[i+1+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ux_w[i+1+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.ux_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / (2. * ((*gpu_def).hz));
			Tz2 = (DevArraysPtr.ro_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.uy_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)] * DevArraysPtr.uy_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)]) / (2. * ((*gpu_def).hz));
		}

		divgrad1 += ((*gpu_def).m[media] * ((*gpu_def).l_w) * ((*gpu_def).c) / 2.) *
		((DevArraysPtr.ro_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - DevArraysPtr.S_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)]) - 2 * DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - S_n) + DevArraysPtr.ro_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - DevArraysPtr.S_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)])) / (((*gpu_def).hx) * ((*gpu_def).hx)) +
		(DevArraysPtr.ro_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * (1 - DevArraysPtr.S_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)]) - 2 * DevArraysPtr.ro_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (1 - S_n) + DevArraysPtr.ro_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * (1 - DevArraysPtr.S_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)])) / (((*gpu_def).hy) * ((*gpu_def).hy)));

		divgrad2 += ((*gpu_def).m[media] * ((*gpu_def).l_n) * ((*gpu_def).c) / 2.) *
		((DevArraysPtr.ro_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.S_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - 2 * DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * S_n + DevArraysPtr.ro_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.S_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (((*gpu_def).hx) * ((*gpu_def).hx)) +
		(DevArraysPtr.ro_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.S_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - 2 * DevArraysPtr.ro_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] * S_n + DevArraysPtr.ro_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.S_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / (((*gpu_def).hy) * ((*gpu_def).hy)));

		Tx1 = (DevArraysPtr.ro_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ux_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ux_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hx));
		Ty1 = (DevArraysPtr.ro_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.uy_w[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.uy_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hy));
		Tx2 = (DevArraysPtr.ro_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ux_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.ux_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hx));
		Ty2 = (DevArraysPtr.ro_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.uy_n[i+(j+1)*localNx+k*localNx*((*gpu_def).Ny)] - DevArraysPtr.ro_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] * DevArraysPtr.uy_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)]) / (2 * ((*gpu_def).hy));

		if (t < 2 * ((*gpu_def).dt))
		{
			A1 = roS_w + (((*gpu_def).dt) / (*gpu_def).m[media]) * (divgrad1 - Tx1 - Ty1 - Tz1);
			A2 = roS_n + (((*gpu_def).dt) / (*gpu_def).m[media]) * (divgrad2 - Tx2 - Ty2 - Tz2);
		}
		else
		{
			A1 = (2 * ((*gpu_def).dt) * ((*gpu_def).dt)) / ((*gpu_def).m[media] * (((*gpu_def).dt) + 2 * ((*gpu_def).tau))) * (divgrad1 - Tx1 - Ty1 - Tz1 + (2 * roS_w * (*gpu_def).m[media] * ((*gpu_def).tau)) / (((*gpu_def).dt) * ((*gpu_def).dt)) + DevArraysPtr.roS_w_old[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (*gpu_def).m[media] * (((*gpu_def).dt) - 2 * ((*gpu_def).tau)) / (2 * ((*gpu_def).dt) * ((*gpu_def).dt)));
			A2 = (2 * ((*gpu_def).dt) * ((*gpu_def).dt)) / ((*gpu_def).m[media] * (((*gpu_def).dt) + 2 * ((*gpu_def).tau))) * (divgrad2 - Tx2 - Ty2 - Tz2 + (2 * roS_n * (*gpu_def).m[media] * ((*gpu_def).tau)) / (((*gpu_def).dt) * ((*gpu_def).dt)) + DevArraysPtr.roS_n_old[i+j*localNx+k*localNx*((*gpu_def).Ny)] * (*gpu_def).m[media] * (((*gpu_def).dt) - 2 * ((*gpu_def).tau)) / (2 * ((*gpu_def).dt) * ((*gpu_def).dt)));
		}

		DevArraysPtr.roS_w_old[local] = roS_w;
		DevArraysPtr.roS_n_old[local] = roS_n;
		DevArraysPtr.roS_w[local] = A1;
		DevArraysPtr.roS_n[local] = A2;
	}
}

// ������ ro*S �� ���� ������ �����
void roS_calculation(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, consts def, double t, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ)
{

	#ifdef NR
	assign_roS_kernel_nr<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,t); 
	#else
	assign_roS_kernel<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,t);
	#endif
	checkErrors("assign roS");
}

// ����� ������� ��� ������ ����� ����� (���������� �� ��������� �����)
__global__ void Newton_method_kernel(ptr_Arrays DevArraysPtr, int localNx) 
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;

	if ((i<localNx-1) && (j<(*gpu_def).Ny-1) && (k<(*gpu_def).Nz) && (i!=0) && (j!=0) && (((k!=0) && (k!=(*gpu_def).Nz-1)) || ((*gpu_def).Nz<2)))
    {
		int media = DevArraysPtr.media[i+j*localNx+k*localNx*((*gpu_def).Ny)];
        double S_e, P_k, AAA, F1, F2, PkS, F1P, F2P, F1S, F2S, det;
		double S_n=DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];
		double P_w=DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)];

        S_e = (1 - S_n - (*gpu_def).S_wr[media]) / (1 - (*gpu_def).S_wr[media]);
        P_k = (*gpu_def).P_d[media] * pow(S_e, -1 / (*gpu_def).lambda[media]);
        AAA = pow(S_e, ((-1 / (*gpu_def).lambda[media]) - 1));
        F1 = ro0_w * (1 + ((*gpu_def).beta_w) * (P_w - P_atm)) * (1 - S_n) - DevArraysPtr.roS_w[i+j*localNx+k*localNx*((*gpu_def).Ny)];
        F2 = ro0_n * (1 + ((*gpu_def).beta_n) * (P_w + P_k - P_atm)) * S_n - DevArraysPtr.roS_n[i+j*localNx+k*localNx*((*gpu_def).Ny)];

        PkS = AAA * (*gpu_def).P_d[media] / ((*gpu_def).lambda[media] * (1 - (*gpu_def).S_wr[media]));
        F1P = ro0_w * ((*gpu_def).beta_w) * (1 - S_n);
        F2P = ro0_n * ((*gpu_def).beta_n) * S_n;
        F1S = (-1) * ro0_w * (1 + ((*gpu_def).beta_w) * (P_w - P_atm));
        F2S = ro0_n * (1 + ((*gpu_def).beta_n) * (P_w + P_k - P_atm + (S_n * PkS)));

        det = F1P * F2S - F1S * F2P;

        DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = P_w - (1 / det) * (F2S * F1 - F1S * F2);
        DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = S_n - (1 / det) * (F1P * F2 - F2P * F1);
    }
}

// ������ �������� ���� P1 � ������������ NAPL S2 �� ���� ������ �����
void Pw_Sn_calculation(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, consts def, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ)
{
	Newton_method_kernel<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx); 
	checkErrors("assign Pw and Sn");
}

// ��������� ������� �� S2
__global__ void Sn_boundary_kernel(ptr_Arrays DevArraysPtr, int localNx, int rank, int size) 
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;

	if ((i<localNx) && (j<((*gpu_def).Ny)) && (k<((*gpu_def).Nz)) && (device_is_active_point(i, localNx, rank, size)==1))
	{
		if ((i == 0) && (((*gpu_def).Nx)>2))
		{
		   DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S_n[i+1+j*localNx+k*localNx*((*gpu_def).Ny)];
		}

		if ((i == localNx - 1) && (((*gpu_def).Nx)>2))
		{
			DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S_n[i-1+j*localNx+k*localNx*((*gpu_def).Ny)];
		}

		if ((j == ((*gpu_def).Ny) - 1) && (((*gpu_def).Ny)>2))
		{
			DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S_n[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)];
		}

		if ((j==0) && (((*gpu_def).Ny)>2))
		{
			int I=device_i_to_I(i,rank,size, gpu_def[0]);
			if ((I>=((*gpu_def).Nx)/2-((*gpu_def).source)) && (I<=((*gpu_def).Nx)/2+((*gpu_def).source)) && (k>=((*gpu_def).Nz)/2-((*gpu_def).source)) && (k<=((*gpu_def).Nz)/2+((*gpu_def).source)))
				DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = S_n_gr;
			else
				DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = 0;
		}

		if ((k == 0) && (((*gpu_def).Nz)>2))
		{
			DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S_n[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)];
		}

		if ((k == ((*gpu_def).Nz) - 1) && (((*gpu_def).Nz)>2))
		{
			DevArraysPtr.S_n[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.S_n[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)];
		}
	}
}

// ��������� ������� �� P1
__global__ void Pw_boundary_kernel(ptr_Arrays DevArraysPtr, int localNx, int rank, int size) 
{
	int blockIdxz=blockIdx.y / BlockNY;
	int blockIdxy=blockIdx.y % BlockNY;
	int i=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.z+blockIdxz*blockDim.z;
	int j=threadIdx.y+blockIdxy*blockDim.y;

	if ((i<localNx) && (j<((*gpu_def).Ny)) && (k<((*gpu_def).Nz)) && (device_is_active_point(i, localNx, rank, size)==1))
	{
		if ((i == 0) && (((*gpu_def).Nx)>2))
		{
			DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P_w[i+1+j*localNx+k*localNx*((*gpu_def).Ny)]; 
		}

		if ((i == localNx - 1) && (((*gpu_def).Nx)>2))
		{
			DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P_w[i-1+j*localNx+k*localNx*((*gpu_def).Ny)];
		}

		if ((j == ((*gpu_def).Ny) - 1) && (((*gpu_def).Ny)>2))
		{
			DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P_w[i+(j-1)*localNx+k*localNx*((*gpu_def).Ny)] + DevArraysPtr.ro_w[i+localNx*1] * g_const * ((*gpu_def).hy);
		}

		if ((j==0) && (((*gpu_def).Ny)>2))
		{
			DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = P_atm;
		}

		if ((k == 0) && (((*gpu_def).Nz)>2))
		{
			DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P_w[i+j*localNx+(k+1)*localNx*((*gpu_def).Ny)]; 
		}

		if ((k == ((*gpu_def).Nz) - 1) && (((*gpu_def).Nz)>2))
		{
			DevArraysPtr.P_w[i+j*localNx+k*localNx*((*gpu_def).Ny)] = DevArraysPtr.P_w[i+j*localNx+(k-1)*localNx*((*gpu_def).Ny)];
		}
	}
}

// ���������� ��������� �������
void boundary_conditions(ptr_Arrays HostArraysPtr, ptr_Arrays DevArraysPtr, int localNx, int rank, int size, int blocksX, int blocksY, int blocksZ, consts def)
{
	Sn_boundary_kernel<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,rank,size); 
	checkErrors("assign Sn");
	Pw_boundary_kernel<<<dim3(blocksX,blocksY*blocksZ), dim3(BlockNX,BlockNY,BlockNZ)>>>(DevArraysPtr,localNx,rank,size); 
	checkErrors("assign Pw");
}	

// ������� �������� ������ � ������ �����
void load_data_to_host(double* HostArrayPtr, double* DevArrayPtr, int localNx, consts def)
{
	hipMemcpy( HostArrayPtr, DevArrayPtr, localNx*(def.Ny)*(def.Nz)*sizeof(double), hipMemcpyDeviceToHost );
	checkErrors("copy data to host");
}

// ������� �������� ������ ���� double � ������ ����������
void load_data_to_device(double* HostArrayPtr, double* DevArrayPtr, int localNx, consts def)
{
	hipMemcpy( DevArrayPtr, HostArrayPtr, localNx*(def.Ny)*(def.Nz)*sizeof(double), hipMemcpyHostToDevice );
	checkErrors("copy double data to device");
}

// ������� �������� ������ ���� int � ������ ����������
void load_data_to_device_int(int* HostArrayPtr, int* DevArrayPtr, int localNx, consts def)
{
	hipMemcpy( DevArrayPtr, HostArrayPtr, localNx*(def.Ny)*(def.Nz)*sizeof(int), hipMemcpyHostToDevice );
	checkErrors("copy int data to device");
}

// ��������� ������ ���������� ��� ������ ����� ��������� �������
void device_memory_allocation(ptr_Arrays* ArraysPtr, double** DevBuffer, int localNx, consts def)
{
	hipMalloc((void**) DevBuffer,  2 * (def.Ny) * (def.Nz) * sizeof(double));

	hipMalloc((void**) &((*ArraysPtr).x),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).y),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).z),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).P_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).P_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).S_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).ro_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).ro_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).ux_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).uy_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).uz_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).ux_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).uy_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).uz_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).Xi_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).Xi_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).roS_w),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).roS_n),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).roS_w_old),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).roS_n_old),  localNx * (def.Ny) * (def.Nz) * sizeof(double));
	hipMalloc((void**) &((*ArraysPtr).media),  localNx * (def.Ny) * (def.Nz) * sizeof(int));

	checkErrors("memory allocation");
}

// ����������� ������ ���������� �� ��� ������� ����� ��������� �������
void device_memory_free(ptr_Arrays DevArraysPtr, double* DevBuffer)
{
	hipFree (DevBuffer);
	hipFree (DevArraysPtr.x);
	hipFree (DevArraysPtr.y);
	hipFree (DevArraysPtr.z);
	hipFree (DevArraysPtr.P_w);
	hipFree (DevArraysPtr.P_n);
	hipFree (DevArraysPtr.S_n);
	hipFree (DevArraysPtr.ro_w);
	hipFree (DevArraysPtr.ro_n);
	hipFree (DevArraysPtr.ux_w);
	hipFree (DevArraysPtr.uy_w);
	hipFree (DevArraysPtr.uz_w);
	hipFree (DevArraysPtr.ux_n);
	hipFree (DevArraysPtr.uy_n);
	hipFree (DevArraysPtr.uz_n);
	hipFree (DevArraysPtr.Xi_w);
	hipFree (DevArraysPtr.Xi_n);
	hipFree (DevArraysPtr.roS_w);
	hipFree (DevArraysPtr.roS_n);
	hipFree (DevArraysPtr.roS_w_old);
	hipFree (DevArraysPtr.roS_n_old);
	hipFree (DevArraysPtr.media);

	checkErrors("memory reliase");
}

// ������������� ����������
// ������ ���������� �� ����������, ����� �������� �����
// ������ ������������ ����������
void device_initialization(int rank, int* blocksX, int* blocksY, int* blocksZ, int localNx, consts def)
{
	// ���� 3 ���������� �� ����� ���� � ������� ����������� ����
	int device=rank%3;
	hipSetDevice(device);
	printf("CUDA initialized.\n");

	// ���������� ����������� ������
	// ���� ����� ����� ����� �� ������ ������� �����,
	// �� ���������� ������ ����� �� 1 ������.
	*blocksX=localNx/BlockNX;
	if (localNx%BlockNX!=0)
		(*blocksX)++;
	*blocksY=(def.Ny)/BlockNY;
	if ((def.Ny)%BlockNY!=0)
		(*blocksY)++;
	*blocksZ=(def.Nz)/BlockNZ;
	if ((def.Nz)%BlockNZ!=0)
		(*blocksZ)++;

	hipMemcpyToSymbol(HIP_SYMBOL( gpu_def), &def, sizeof ( consts ), 0, hipMemcpyHostToDevice );
	checkErrors("constant memory copy");

	int deviceCount;
    hipDeviceProp_t devProp;
    hipGetDeviceCount ( &deviceCount );

        hipGetDeviceProperties ( &devProp, device );
        printf ( "Device %d\n", device );
        printf ( "Compute capability : %d.%d\n", devProp.major, devProp.minor );
        printf ( "Name : %s\n", devProp.name );
        printf ( "Total Global Memory : %ld\n", devProp.totalGlobalMem );
        printf ( "Shared memory per block: %d\n", devProp.sharedMemPerBlock );
        printf ( "Registers per block : %d\n", devProp.regsPerBlock );
        printf ( "Warp size : %d\n", devProp.warpSize );
        printf ( "Max threads per block : %d\n", devProp.maxThreadsPerBlock );
        printf ( "Total constant memory : %d\n\n", devProp.totalConstMem );

		// ������������ ������ ��������� ����� ��� ����������
		// 21 - ���������� ���������� � �����
		printf ( "Total NAPL_Filtration grid size : %d\n\n", devProp.totalGlobalMem/(21*sizeof(double)) );

		// localNX+2 ������ ��� 2NyNz �� ����� ������ ����������
		if ((localNx+2)*(def.Ny)*(def.Nz) > (devProp.totalGlobalMem/(21*sizeof(double))))
			printf ("\nError! Not enough memory at GPU, rank=%d\n",rank);
}


__global__ void load_exchange_data_kernel(double* DevArrayPtr, double* DevBuffer, int localNx)
{
	int j=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.y+blockIdx.y*blockDim.y;

	if (j<(*gpu_def).Ny && k<(*gpu_def).Nz)
	{
		DevBuffer[j+((*gpu_def).Ny)*k]=DevArrayPtr[1+localNx*j+localNx*((*gpu_def).Ny)*k];
		DevBuffer[j+((*gpu_def).Ny)*k+((*gpu_def).Ny)*((*gpu_def).Nz)]=DevArrayPtr[localNx-2+localNx*j+localNx*((*gpu_def).Ny)*k];
	}
}


void load_exchange_data(double* HostArrayPtr, double* DevArrayPtr, double* HostBuffer, double* DevBuffer, int localNx, int blocksY, int blocksZ, int rank, int size, consts def)
{
	load_exchange_data_kernel<<<dim3(blocksY,blocksZ), dim3(BlockNY,BlockNZ)>>>(DevArrayPtr, DevBuffer, localNx); 
	checkErrors("load_exchange_data");
	hipMemcpy( HostBuffer, DevBuffer, 2*(def.Ny)*(def.Nz)*sizeof(double), hipMemcpyDeviceToHost );
	checkErrors("copy data to host");
}

__global__ void save_exchange_data_kernel(double* DevArrayPtr, double* DevBuffer, int localNx, int rank, int size)
{
	int j=threadIdx.x+blockIdx.x*blockDim.x;
	int k=threadIdx.y+blockIdx.y*blockDim.y;

	if (j<(*gpu_def).Ny && k<(*gpu_def).Nz)
	{
		if (rank!=0)
			DevArrayPtr[localNx*j+localNx*((*gpu_def).Ny)*k]=DevBuffer[j+((*gpu_def).Ny)*k];
		if (rank!=size-1)
			DevArrayPtr[localNx-1+localNx*j+localNx*((*gpu_def).Ny)*k]=DevBuffer[j+((*gpu_def).Ny)*k+((*gpu_def).Ny)*((*gpu_def).Nz)];
	}
}

void save_exchange_data(double* HostArrayPtr, double* DevArrayPtr, double* HostBuffer, double* DevBuffer, int localNx, int blocksY, int blocksZ, int rank, int size, consts def)
{
	hipMemcpy( DevBuffer, HostBuffer, 2*(def.Ny)*(def.Nz)*sizeof(double), hipMemcpyHostToDevice );
	checkErrors("copy data to device");
	save_exchange_data_kernel<<<dim3(blocksY,blocksZ), dim3(BlockNY,BlockNZ)>>>(DevArrayPtr, DevBuffer, localNx, rank, size); 
	checkErrors("save_exchange_data");
}